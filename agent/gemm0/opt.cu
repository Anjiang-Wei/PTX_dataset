#include "hip/hip_runtime.h"
// opt.cu
// Optimized SGEMM for A100 (sm_80) with 128x128x8 tiling.
// Must match the declaration: extern __global__ void sgemm_optimized(...)

__global__
void sgemm_optimized(const float* __restrict__ A,
                     const float* __restrict__ B,
                     float* __restrict__ C) {
    constexpr int TILE_M = 128;
    constexpr int TILE_N = 128;
    constexpr int TILE_K = 8;
    constexpr int BLK_X  = 32;
    constexpr int BLK_Y  = 8;
    constexpr int RM     = TILE_M / BLK_Y;  // 16
    constexpr int RN     = TILE_N / BLK_X;  // 4

    __shared__ float As[TILE_M][TILE_K];
    __shared__ float Bs[TILE_K][TILE_N];

    int tx = threadIdx.x, ty = threadIdx.y;
    int block_m = blockIdx.y * TILE_M;
    int block_n = blockIdx.x * TILE_N;

    int row0 = block_m + ty * RM;
    int col0 = block_n + tx * RN;

    float acc[RM][RN] = {0};

    int num_tiles = (K + TILE_K - 1) / TILE_K;
    int tid = ty * BLK_X + tx;
    int THR = BLK_X * BLK_Y;

    for (int t = 0; t < num_tiles; ++t) {
        int aCol = t * TILE_K;
        for (int idx = tid; idx < TILE_M * TILE_K; idx += THR) {
            int i = idx / TILE_K, k = idx % TILE_K;
            int gRow = block_m + i, gCol = aCol + k;
            As[i][k] = (gRow < M && gCol < K) ? A[gRow * K + gCol] : 0.0f;
        }

        int bRow = t * TILE_K;
        for (int idx = tid; idx < TILE_K * TILE_N; idx += THR) {
            int k = idx / TILE_N, j = idx % TILE_N;
            int gRow = bRow + k, gCol = block_n + j;
            Bs[k][j] = (gRow < K && gCol < N) ? B[gRow * N + gCol] : 0.0f;
        }

        __syncthreads();

        #pragma unroll
        for (int kk = 0; kk < TILE_K; ++kk) {
            float aFrag[RM], bFrag[RN];
            #pragma unroll
            for (int i = 0; i < RM; ++i)
                aFrag[i] = As[ty * RM + i][kk];
            #pragma unroll
            for (int j = 0; j < RN; ++j)
                bFrag[j] = Bs[kk][tx * RN + j];
            #pragma unroll
            for (int i = 0; i < RM; ++i)
                for (int j = 0; j < RN; ++j)
                    acc[i][j] += aFrag[i] * bFrag[j];
        }
        __syncthreads();
    }

    #pragma unroll
    for (int i = 0; i < RM; ++i) {
        int r = row0 + i;
        if (r >= M) continue;
        #pragma unroll
        for (int j = 0; j < RN; ++j) {
            int c = col0 + j;
            if (c >= N) continue;
            float val = ALPHA * acc[i][j];
            if (BETA != 0.0f) val += BETA * C[r * N + c];
            C[r * N + c] = val;
        }
    }
}
