
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime.h>

#define M 2048
#define N 2048
#define K 2048
#define ALPHA 1.0f
#define BETA 0.0f

// Launch config for optimized kernel (must match opt.cu)
#define OPT_TILE_M 128
#define OPT_TILE_N 128
#define OPT_THREADS_X 32
#define OPT_THREADS_Y 8


    extern __global__ void sgemm_global_mem_coalesce(const float*,const float*,float*);
    extern __global__ void sgemm_optimized(const float*,const float*,float*);
    

#define CUDA_CHECK(x) do { hipError_t e=x; if(e!=hipSuccess){   fprintf(stderr,"CUDA error %s:%d: %s\n",__FILE__,__LINE__,hipGetErrorString(e)); exit(1);} }while(0)

// Baseline launcher: 1D block of BLOCKSIZE*BLOCKSIZE threads
template <typename F>
void launch_baseline(F kernel, const char* name, float* dA, float* dB, float* dC, int BLOCKSIZE) {
  dim3 grid((M+BLOCKSIZE-1)/BLOCKSIZE, (N+BLOCKSIZE-1)/BLOCKSIZE);
  dim3 block(BLOCKSIZE * BLOCKSIZE);
  printf("Launching %s with grid=(%d,%d) block=(%d)\n",
         name, grid.x, grid.y, block.x);
  kernel<<<grid, block>>>(dA, dB, dC);
  CUDA_CHECK(hipDeviceSynchronize());
}

// Optimized launcher: 32x8 threads with 16x4 register blocking, 128x128 tiles
template <typename F>
void launch_optimized(F kernel, const char* name, float* dA, float* dB, float* dC) {
  dim3 grid((N + OPT_TILE_N - 1) / OPT_TILE_N,
            (M + OPT_TILE_M - 1) / OPT_TILE_M);
  dim3 block(OPT_THREADS_X, OPT_THREADS_Y);
  printf("Launching %s with grid=(%d,%d) block=(%d,%d) tile=(%d,%d)\n",
         name, grid.x, grid.y, OPT_THREADS_X, OPT_THREADS_Y, OPT_TILE_M, OPT_TILE_N);
  kernel<<<grid, block>>>(dA, dB, dC);
  CUDA_CHECK(hipDeviceSynchronize());
}

int main(){
  printf("Comparing kernels on sm_80\n");
  size_t szA=M*K*sizeof(float), szB=K*N*sizeof(float), szC=M*N*sizeof(float);
  float *hA=(float*)malloc(szA),*hB=(float*)malloc(szB);
  for(int i=0;i<M*K;++i)hA[i]=(float)rand()/RAND_MAX;
  for(int i=0;i<K*N;++i)hB[i]=(float)rand()/RAND_MAX;

  float *dA,*dB,*dC1,*dC2;
  CUDA_CHECK(hipMalloc(&dA,szA)); CUDA_CHECK(hipMalloc(&dB,szB));
  CUDA_CHECK(hipMalloc(&dC1,szC)); CUDA_CHECK(hipMalloc(&dC2,szC));
  CUDA_CHECK(hipMemcpy(dA,hA,szA,hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(dB,hB,szB,hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemset(dC1,0,szC)); CUDA_CHECK(hipMemset(dC2,0,szC));

  // Launch baseline (BLOCKSIZE*BLOCKSIZE threads)
  launch_baseline(sgemm_global_mem_coalesce, "baseline", dA, dB, dC1, 32);

  // Launch optimized (32x8 threads with 16x4 register blocking, 128x128 tiles)
  launch_optimized(sgemm_optimized, "optimized", dA, dB, dC2);

  float *hC1=(float*)malloc(szC),*hC2=(float*)malloc(szC);
  CUDA_CHECK(hipMemcpy(hC1,dC1,szC,hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(hC2,dC2,szC,hipMemcpyDeviceToHost));

  // Relative L2 difference
  double diff=0,ref=0;
  for(int i=0;i<M*N;++i){double a=hC1[i],b=hC2[i];diff+=(a-b)*(a-b);ref+=a*a;}
  diff=sqrt(diff/(ref+1e-12));
  printf("Relative L2 diff = %.6e\n",diff);
  printf(diff<1e-4?"Equivalent.\n":"Different!\n");

  free(hA);free(hB);free(hC1);free(hC2);
  CUDA_CHECK(hipFree(dA));CUDA_CHECK(hipFree(dB));CUDA_CHECK(hipFree(dC1));CUDA_CHECK(hipFree(dC2));
  return 0;
}
